#include <iostream>
#include <hip/hip_runtime.h>


using namespace std;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
__global__ void AddInts(int* a, int*b)
{
    a[0] += b[0];
}
int main()
{
    int a = 5;
    int b = 9;

    int *d_a, *d_b;

    hipMalloc(&d_a, sizeof(int));
    hipMalloc(&d_b, sizeof(int));

    hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);

    AddInts<<<1, 1>>>(d_a, d_b);

    gpuErrchk(hipPeekAtLastError());

    hipMemcpy(&a, d_a, sizeof(int), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    cout << "The answer is " << a << endl;

    hipFree(d_a);
    hipFree(d_b);

    return 0;
}