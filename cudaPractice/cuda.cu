
#include <hip/hip_runtime.h>
#include  <iostream>
#include <stdio.h>


#define gpuMac(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
    //printf(cudaGetErrorString(code));
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
__global__ void VectorFake(hipPitchedPtr a, hipPitchedPtr b, hipPitchedPtr ret, int count)
{
    if(threadIdx.x == 49 && threadIdx.y == 49 && threadIdx.z == 49)
    {
    }
    return;
}
__global__ void fake(float* b, int count)
{
    b[count - 1] = 0.2f;
}
__global__ void VecAdd(float* A, float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}
__global__
void add(int n, float *x, float *y)
{
  for (int i = 0; i < n; i++)
    y[i] = x[i] + y[i];
}
extern "C" void callCuda(float*** a, float*** b, float*** ret, int count);

void callCuda(float*** a, float*** b, float*** ret, int count)//=>count is dimesnion of 3d cube
{
    // //assume that the size of a,b, ret are equal to count * count * count


    // //I accidentally deleted this file with a move operation, it will now be simpler and worse

    // cudaExtent extent{count * sizeof(float), count, count};

    // //long allocation process, hope it works lol
    // cudaPitchedPtr cudaA;
    // cudaPitchedPtr cudaB;
    // cudaPitchedPtr cudaRet;
    // cudaMalloc3D(&cudaA, extent);
    // cudaMalloc3D(&cudaB, extent);
    // cudaMalloc3D(&cudaRet, extent);

    // cudaMemcpy3DParms cudaAParms = {0};
    // cudaMemcpy3DParms cudaBParms = {0};

    // cudaAParms.srcPtr = make_cudaPitchedPtr(a, extent.width, extent.depth, extent.height);
    // cudaAParms.dstPtr = cudaA;
    // cudaAParms.extent = extent;
    // cudaAParms.kind = cudaMemcpyHostToDevice;

    // cudaAParms.srcPtr = make_cudaPitchedPtr(b, extent.width, extent.depth, extent.height);
    // cudaAParms.dstPtr = cudaB;
    // cudaAParms.extent = extent;
    // cudaAParms.kind = cudaMemcpyHostToDevice;

    // cudaMemcpy3D(&cudaAParms);
    // cudaMemcpy3D(&cudaBParms);

    // dim3 threadDim(8,8,8);
    // //thread blocks can support a maximum of 1024 threads at once, so the
    // //cube processing has to be constrained to these dimesnions
    // //processing with 8,8,8 as that is maximum even cube within 1024 that is
    // //a multiple of 2


    // dim3 cubeDim((count + threadDim.x - 1)/ threadDim.x, (count + threadDim.y - 1) / threadDim.y, (count + threadDim.z - 1) / threadDim.z);
    // //cuda extent is defined weirldy where the width element is quote:
    // //Width in elements when referring to array memory, in bytes when referring to linear memory
    // //and height and depth are both just refering to elelment count
    // //so width, the first elemenet must be multiplied by the variable size

    // VectorFake<<<cubeDim, threadDim>>>(cudaA, cudaB, cudaRet, count);

    // ret = (float***)cudaRet.ptr;//hopefully this works

    // std::cout << "HI FROM GPU!" << std::endl;

    // float* e;

    // cudaMalloc(&e, sizeof(float) * count);
    // cudaMemcpy(e, a[count - 1][count - 1], sizeof(float) * count, cudaMemcpyHostToDevice);

    // fake<<<1, 1>>>(e, count);

    // cudaDeviceSynchronize();

    // cudaMemcpy(a[count - 1][count - 1], e, sizeof(float)* count, cudaMemcpyDeviceToHost);

    // cudaDeviceSynchronize();

    // std::cout << a[count - 1][count - 1][count - 1] << std::endl;
    // //std::cout << e[0] << std::endl;

    int N = 1<<20;
    float *x, *y;

    hipDeviceSynchronize();

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // Run kernel on 1M elements on the GPU
    add<<<1, 1>>>(N, x, y);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < 100; i++)
        maxError = fmax(maxError, fabs(y[i]-3.0f));
    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);

    return;
}